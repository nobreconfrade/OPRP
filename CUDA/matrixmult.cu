#include "hip/hip_runtime.h"
#include <iostream>

__global__ void matrixMulKernel (float* M, float* N, float* P, int Width){
  int Row = blockIdx.y*blockDim.y+threadIdx.y;
  int Col = blockIdx.x*blockDim.x+threadIdx.x;
  if ((Row < Width) && (Col < Width)) {
    float Pvalue = 0;
    for (int k = 0; k < Width; ++k) {
      Pvalue += M[Ŕow*Width+k]*N[k*Width+Col];
    }
    P[Row*Width+Col] = Pvalue;
  }
}

void dados(float *M, int size)
{
	int i, j;
	srand(time(NULL));
	for (int i = 0; i < size * size; ++i) {
		M[i] = rand();
	}
}

int main(int argc, char const *argv[]) {
  int size;
  size = atoi(argv[1]);

  float MCPU[size][size], NCPU[size][size], PCPU[size][size];
  dados(MCPU,size);
  dados(NCPU,size);

  float *M, *N, *P;
  hipMalloc((void**)&M, size * size * sizeof(float));
  hipMalloc((void**)&N, size * size * sizeof(float));
  hipMalloc((void**)&P, size * size * sizeof(float));

  hipMemcpy(M, MCPU, size * size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(N, NCPU, size * size * sizeof(float), hipMemcpyHostToDevice);

  dim3 Grade(1,1);
  dim3 Bloco(size,size);

  matrixMulKernel<<Grade,Bloco>>>(M,N,P,size);


  hipMemcpy(PCPU, P, size * size * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      std::cout << PCPU[i][j]<< '\t';
    }
    std::cout << "" << '\n';
  }

  return 0;
}
