
#include <hip/hip_runtime.h>
#include <stdio.h>
__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
  printf("%c\n",STR[threadIdx.x % STR_LENGTH]);
}

int main(void)
{
  int num_threads = STR_LENGTH;
  int num_blocks = 2;
  dim3 dimBlock(num_threads);
  dim3 dimGrid(1,1);
  hello<<<dimGrid,dimBlock>>>();
  hipDeviceSynchronize();
  return 0;
}
