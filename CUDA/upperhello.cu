
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

const int N = 16;
const int CORES = 16;

__global__ void hello(char* s){
  if ((s[blockIdx.x] >= 'a')&&(s[blockIdx.x] <= 'z')) {
    s[blockIdx.x] -= 32;
  }
}

int main(int argc, char const *argv[]) {
  char cpu_string[N] = "hello world!";

  char* gpu_string;

  hipMalloc((void**)&gpu_string, N * sizeof(char));

  hipMemcpy(gpu_string, cpu_string, N * sizeof(char), hipMemcpyHostToDevice);

  hello<<<CORES,1>>>(gpu_string);

  hipMemcpy(cpu_string, gpu_string, N * sizeof(char), hipMemcpyDeviceToHost);

  hipFree(gpu_string);

  cout << cpu_string << endl;

  return 0;
}
